#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>


#define BLOCKS 1000
#define THREADSPERBLOCK 1000
#define size 10

__global__ void incrementVector(int *data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    index = index % size;
    atomicAdd(&data[index],1);
}

int main() {

    int data[1000000];
    int data_out[1000000];
    for(int i = 0; i < 1000000; i++){
        data[i] = i;
    }

    int *d_in;
    hipMalloc((void**)&d_in, 1000000*sizeof(int));
    hipMemcpy(d_in, data, 1000000*sizeof(int), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0); 

    incrementVector<<<1000,1000>>>(d_in);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);


    hipMemcpy(data_out, d_in, 1000000*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int i = 0; i < 10; i++){
        printf("%d--", data_out[i]);
    }
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\nTotal kernel time: %3.2f ms\n", elapsedTime);



}