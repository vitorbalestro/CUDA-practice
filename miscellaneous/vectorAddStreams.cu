#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define N (1024 * 1024)
#define TOTAL_SIZE (N*21)


__global__ void vectorAdd(int *d_a, int *d_b, int *d_c){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    d_c[i] = d_a[i] + d_b[i];
}

int main() {

    int *h_a, *h_b, *h_c;
    hipHostAlloc((void**)&h_a,TOTAL_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_b,TOTAL_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_c,TOTAL_SIZE * sizeof(int), hipHostMallocDefault);    
    
    for(int i = 0; i < TOTAL_SIZE; i++){
        h_a[i] = 0;
        h_b[i] = 1;
    }
    int *d_a1, *d_a2, *d_a3;
    int *d_b1, *d_b2, *d_b3;
    int *d_c1, *d_c2, *d_c3;

    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    for(int i = 0; i < TOTAL_SIZE; i+=N*3){
        hipMemcpyAsync(d_a1, h_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_a2, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d_a3, h_a+i+2*N, N*sizeof(int), hipMemcpyHostToDevice, stream3);

        hipMemcpyAsync(d_b1, h_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_b2, h_b+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d_b3, h_b+i+2*N, N*sizeof(int), hipMemcpyHostToDevice, stream3);

        vectorAdd<<<N/256, 256, 0, stream1>>>(d_a1,d_b1,d_c1);
        vectorAdd<<<N/256, 256, 0, stream2>>>(d_a2,d_b2,d_c2);
        vectorAdd<<<N/256, 256, 0, stream3>>>(d_a3,d_b3,d_c3);

        hipMemcpyAsync(h_c+i, d_c1, N*sizeof(int),hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(h_c+i+N, d_c2, N*sizeof(int),hipMemcpyDeviceToHost, stream2);
        hipMemcpyAsync(h_c+i+2*N, d_c3, N*sizeof(int),hipMemcpyDeviceToHost, stream3);

    }

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);

    //frees...

    for(int i = 0; i < 100; i++) {
        printf("%d--", h_c[i]);
    }
    printf("\n");

}