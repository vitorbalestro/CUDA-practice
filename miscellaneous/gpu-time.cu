#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0); 
    // do work...

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Total GPU time: %3.1f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}