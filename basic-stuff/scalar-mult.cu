#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void scalar_multiplication(int *d_a, int *d_b, int *d_c, int N){
    int i = threadIdx.x;
    while(i < N) {
        d_b[i] = *d_c * d_a[i];
        i += blockDim.x;
    }
}

int main() {

    int c = 10;
    int a[5] = {1,2,3,4,5}, b[5];
    int *d_a, *d_b;
    int *d_c;


    int size = 5 * sizeof(int);


    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void **)&d_c, sizeof(int));

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice);

    scalar_multiplication<<<1,5>>>(d_a,d_b,d_c,5);

    hipMemcpy(&b, d_b, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i = 0; i < 5; i++){
        printf("%d ", b[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    


    return 0;
}