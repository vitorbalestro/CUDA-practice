#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "/usr/include/postgresql/libpq-fe.h"
#include <time.h>
#include <string.h>
#include <sys/time.h>


__global__ void get_coefficients(float *x_vec, float *y_vec, int data_size, float *x_sum, float *x_squared_sum, float *y_sum, float *inner_prod){

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < data_size){
        *x_sum += x_vec[idx];
        *x_squared_sum += x_vec[idx] * x_vec[idx];
        *y_sum += y_vec[idx];
        *inner_prod += x_vec[idx] * y_vec[idx];
    }
}

int main() {

    float *x_sum;
    float *x_squared_sum;
    float *y_sum;
    float *inner_prod;

    hipMallocManaged(&x_sum, sizeof(float));
    hipMallocManaged(&x_squared_sum, sizeof(float));
    hipMallocManaged(&y_sum, sizeof(float));
    hipMallocManaged(&inner_prod,sizeof(float));

    

    return 0;

}