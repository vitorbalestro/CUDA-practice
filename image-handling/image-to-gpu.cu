#include "hip/hip_runtime.h"
// TO COMPILE: nvcc image-to-gpu.cu -o (name of the executable) -ljpeg
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <jpeglib.h>

__host__ int readJPEG(const char* filename, unsigned char** image_data, int* width, int* height, int *num_components) {
    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;

    FILE* infile = fopen(filename, "rb");
    if (!infile) {
        fprintf(stderr, "Error opening JPEG file.\n");
        return 1;
    }

    cinfo.err = jpeg_std_error(&jerr);

    jpeg_create_decompress(&cinfo);

    jpeg_stdio_src(&cinfo, infile);

    jpeg_read_header(&cinfo, TRUE);

    jpeg_start_decompress(&cinfo);

    *width = cinfo.output_width;
    *height = cinfo.output_height;
    *num_components = cinfo.num_components;
    *image_data = (unsigned char*)malloc(*width * *height * *num_components);

    while (cinfo.output_scanline < cinfo.output_height) {
        unsigned char* row = *image_data + cinfo.output_scanline * *width * *num_components;
        jpeg_read_scanlines(&cinfo, &row, 1);
    }

    jpeg_finish_decompress(&cinfo);

    jpeg_destroy_decompress(&cinfo);
    fclose(infile);

    return 0;
}

__host__ void array2JPEG(const char *outputFileName, unsigned char *image_data, int width, int height, int num_components) {
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;

    FILE *outfile;
    if ((outfile = fopen(outputFileName, "wb")) == NULL) {
        fprintf(stderr, "Can't open %s\n", outputFileName);
        return;
    }

    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);
    jpeg_stdio_dest(&cinfo, outfile);

    cinfo.image_width = width;
    cinfo.image_height = height;
    cinfo.input_components = num_components; 
    cinfo.in_color_space = JCS_RGB;

    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, 90, TRUE);

    jpeg_start_compress(&cinfo, TRUE);

    int row_stride = width * num_components;
    JSAMPROW row_pointer;
    while (cinfo.next_scanline < cinfo.image_height) {
        row_pointer = &image_data[cinfo.next_scanline * row_stride];
        jpeg_write_scanlines(&cinfo, &row_pointer, 1);
    }

    jpeg_finish_compress(&cinfo);
    jpeg_destroy_compress(&cinfo);
    fclose(outfile);
}

__global__ void send_image_to_GPU(unsigned char *d_pin, unsigned char *d_pout, int width, int height, int num_components){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < height && col < width * num_components){
        d_pout[col + row * width * num_components] = d_pin[col + row * width * num_components];
    }
}
int main() {

    const char* filename = "JF.jpeg"; // Change to your JPEG file
    const char* output_name = "JF(test).jpeg";
    unsigned char* image_data;
    int width, height,num_components;
    
    
    if (readJPEG(filename, &image_data, &width, &height, &num_components) != 0) {
        fprintf(stderr, "Error reading JPEG image.\n");
        return 1;
    }
    unsigned char *d_pin, *d_pout;
    int size = num_components * width * height;
    unsigned char* output_image;
    output_image = (unsigned char*)malloc(size);

    hipMalloc((void**)&d_pin,size);
    hipMalloc((void**)&d_pout,size);
    hipMemcpy(d_pin, image_data, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock (16,16);
    dim3 gridDim ((height-1)/16 + 1, (width * num_components-1)/16 + 1);

    send_image_to_GPU<<<gridDim, threadsPerBlock>>>(d_pin, d_pout, width, height, num_components);
    
    hipMemcpy(output_image,d_pout,size,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    array2JPEG(output_name,output_image,width,height,num_components);

    free(image_data);
    free(output_image);
    hipFree(d_pin);
    hipFree(d_pout);

    return 0;
}
