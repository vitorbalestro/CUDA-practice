#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void hillis_steele_scan(int *d_data, int n){

    int index = threadIdx.x + blockDim.x*blockIdx.x;
    
    for(int step = 1; step < n; step = 2*step){
        if(index-step >= 0){
            d_data[index] = d_data[index] + d_data[index-step];
        }
        __syncthreads();
    }

}

int main() {

    int data[6] = {1,3,6,2,9,4};
    int *d_data;
    hipMalloc((void**)&d_data,6*sizeof(int));
    hipMemcpy(d_data,&data,6*sizeof(int),hipMemcpyHostToDevice);
   

    hillis_steele_scan<<<1,6>>>(d_data,6);

    hipMemcpy(&data,d_data,6*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int i = 0; i < 6; i++){
        printf("%d--", data[i]);
    }

    hipFree(d_data);

    return 0;
}