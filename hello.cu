#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void my_kernel() {
    printf("Hello from GPU!\n");
}

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    /*my_kernel<<<3,5>>>();
    cudaDeviceSynchronize();
    printf("Hello from CPU!\n");*/

    int a, b, c; // CPU
    int *d_a, *d_b, *d_c; // GPU
    int size = sizeof(int);

    // Allocate space for device
    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_b,size);
    hipMalloc((void**)&d_c,size);

    // Setup input values
    a = 10;
    b = 20;

    // CPU -> GPU
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // kernel execution: 1 thread
    add<<<1,1>>>(d_a,d_b,d_c);
    
    // GPU -> CPU
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    
    // Clean memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("%d\n", c);

    return 0;
}